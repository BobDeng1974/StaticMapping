#include "hip/hip_runtime.h"
#include "icp_cuda.h"

#include <cugar/basic/timer.h>
#include <cugar/kd/cuda/kd_builder.h>
#include <cugar/kd/cuda/kd_context.h>
#include <cugar/kd/cuda/knn.h>
#include <cugar/sampling/random.h>
#include <thrust/gather.h>

#include "common/simple_thread_pool.h"

#define DEBUG_CUDA 0

namespace static_map {
namespace registrator {
namespace cuda {

void init_cuda_device() { hipSetDevice(0); }

using cugar::device_tag;
using cugar::host_tag;
bool knn_cugar(const float *ref, int ref_points_num, const float *query,
               int query_points_num, float *knn_dist2, int *knn_index) {
#if DEBUG_CUDA
  hipEvent_t start, stop;
  float dtime = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
#endif

  cugar::Vector3f bbox_min(1.e6);
  cugar::Vector3f bbox_max(-1.e6);
  cugar::vector<host_tag, cugar::Vector4f> h_ref_points(ref_points_num);
  for (int i = 0; i < ref_points_num; ++i) {
    h_ref_points[i] = cugar::Vector4f(
        ref[i * kPointDim], ref[i * kPointDim + 1], ref[i * kPointDim + 2], 0.);
    if (h_ref_points[i][0] < bbox_min[0]) {
      bbox_min[0] = h_ref_points[i][0];
    }
    if (h_ref_points[i][1] < bbox_min[1]) {
      bbox_min[1] = h_ref_points[i][1];
    }
    if (h_ref_points[i][2] < bbox_min[2]) {
      bbox_min[2] = h_ref_points[i][2];
    }

    if (h_ref_points[i][0] > bbox_max[0]) {
      bbox_max[0] = h_ref_points[i][0];
    }
    if (h_ref_points[i][1] > bbox_max[1]) {
      bbox_max[1] = h_ref_points[i][1];
    }
    if (h_ref_points[i][2] > bbox_max[2]) {
      bbox_max[2] = h_ref_points[i][2];
    }
  }
  cugar::vector<host_tag, cugar::Vector4f> h_query_points(query_points_num);
  for (int i = 0; i < query_points_num; ++i) {
    h_query_points[i] =
        cugar::Vector4f(query[i * kPointDim], query[i * kPointDim + 1],
                        query[i * kPointDim + 2], 1.f);
  }

#if DEBUG_CUDA
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&dtime, start, stop);
  fprintf(stderr, "bbox time: %f ms\n", dtime);
#endif

  cugar::vector<device_tag, cugar::Vector4f> d_ref_points(h_ref_points);
  cugar::vector<device_tag, cugar::Vector4f> d_query_points(h_query_points);
  cugar::vector<device_tag, cugar::Kd_node> kd_nodes;
  cugar::vector<device_tag, uint2> kd_leaves;
  cugar::vector<device_tag, uint2> kd_ranges;
  cugar::vector<device_tag, cugar::uint32> kd_index;

  cugar::cuda::Kd_context context(&kd_nodes, &kd_leaves, &kd_ranges);
  cugar::cuda::Kd_builder<cugar::uint32> builder;

// build kdtree
#if DEBUG_CUDA
  hipEventRecord(start, 0);
#endif
  builder.build(context, kd_index, cugar::Bbox3f(bbox_min, bbox_max),
                d_ref_points.begin(), d_ref_points.end(), 8u);
#if DEBUG_CUDA
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&dtime, start, stop);
  fprintf(stderr, "build time: %f ms\n", dtime);
#endif

  cugar::vector<device_tag, cugar::Vector4f> d_sorted_points(ref_points_num);
  thrust::gather(kd_index.begin(), kd_index.begin() + ref_points_num,
                 d_ref_points.begin(), d_sorted_points.begin());
  d_ref_points = d_sorted_points;

#if DEBUG_CUDA
  hipEventRecord(start, 0);
#endif
  // knn search
  cugar::cuda::Kd_knn<kPointDim> knn;
  cugar::vector<device_tag, cugar::cuda::Kd_knn_result> d_results(
      query_points_num * kNNForNormal);
  const cugar::Vector4f *kd_points_ptr = raw_pointer(d_ref_points);
  const cugar::Vector4f *query_points_ptr = raw_pointer(d_query_points);
  knn.run<kNNForNormal>(
      query_points_ptr, query_points_ptr + query_points_num,
      raw_pointer(kd_nodes), raw_pointer(kd_ranges), raw_pointer(kd_leaves),
      cugar::cuda::make_load_pointer<cugar::cuda::LOAD_LDG>(kd_points_ptr),
      raw_pointer(d_results));

#if DEBUG_CUDA
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&dtime, start, stop);
  fprintf(stderr, "search time: %f ms\n", dtime);
#endif

  // cugar::cuda::sync_and_check_error("knn search error.");
  if (knn_dist2 && knn_index) {
    cugar::vector<host_tag, cugar::cuda::Kd_knn_result> h_results(d_results);
#pragma omp parallel for num_threads(LOCAL_OMP_THREADS_NUM)
    for (int i = 0; i < kNNForNormal; ++i) {
      for (int j = 0; j < query_points_num; ++j) {
        const int index = j * kNNForNormal + i;
        knn_dist2[index] = h_results[index].dist2;
        knn_index[index] = h_results[index].index;
      }
    }
  }

#if DEBUG_CUDA
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif

  return true;
}

}  // namespace cuda
}  // namespace registrator
}  // namespace static_map