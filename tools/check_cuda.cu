// MIT License

// Copyright (c) 2019 Edward Liu

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.


#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
  hipDeviceProp_t dP;
  float min_cc = 3.0;

  int rc = hipGetDeviceProperties(&dP, 0);
  if (rc != hipSuccess) {
    hipError_t error = hipGetLastError();
    printf("CUDA error: %s", hipGetErrorString(error));
    return rc; /* Failure */
  }
  if ((dP.major + (dP.minor / 10)) < min_cc) {
    printf(
        "Min Compute Capability of %2.1f required:  %d.%d found\n Not Building "
        "CUDA Code",
        min_cc, dP.major, dP.minor);
    return 1; /* Failure */
  } else {
    printf("-arch=sm_%d%d", dP.major, dP.minor);
    return 0; /* Success */
  }
}
